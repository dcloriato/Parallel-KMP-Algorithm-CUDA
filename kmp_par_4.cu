#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "timerc.h"

//Author: Dario Loriato
//Apologies for the multiple print statements within the functions, there was a lot of testing to be done


__global__ void alg(char* text_string, char* word, int word_len, int pat_len, int wrd_tbl[], int ans[]){
	int ix=blockIdx.x*blockDim.x+threadIdx.x;
	int pos=ix*128;
        int j=0;
	
	//TESTING THE PARALLEL ALG
	//printf("Thread ID: %d\n",ix);
	//printf("This is the word: %s\n", word);
	//printf("Word len: %d, Pat len: %d\n", word_len, pat_len);
	//printf("This is the word table: ");
	//for(int i=0;i<word_len;i++){
	//	printf("%d",wrd_tbl[i]);
	//}
	//printf("\n");
	//printf("Printing out the characters for text_string: ");
	//for(int i=0;i<10;i++){
	//	printf("<%c>", text_string[i]);
	//}
	//printf("\n");

	for(int i=0;i<128;i++){
		//printf("Index is: %d, Char is: %c\n", pos, text_string[pos]);
		if(text_string[pos]==word[j+1]){
			j++;
			//printf("Char in text: %c Matches with char: %c WrdTbl value is: %d Index is: %d \n", text_string[pos], word[j+1],j,i);
		}else{
			j=wrd_tbl[j];
		}
		if((j+1)==word_len){
			//printf("Pattern match found at position %d \n", (pos-(word_len-2)));
			ans[pos-(word_len-2)]=1;
			j=0;
		}
		ans[pos]=0;
		pos++;
	}
	if(j>0){
		ans[pos-j]=2;
	}
}

void serial_alg(char* text_string, char* word, int word_len, int pat_len, int wrd_tbl[]){
	int j=0;
	for(int i=0;i<pat_len;i++){
		if(text_string[i]==word[j+1]){
			j++;
		}else{
			j=wrd_tbl[j];
		}
		if((j+1)==word_len){
			printf("Pattern match found at position %d\n", (i-(word_len-2)));
		}
	}
}

void tableBuild(char* word, int wrd_tbl[]){
	int k = strlen(word);
	//printf("Length of string %d\n",k);
	int match;
	for(int i=1;i<(k-1);i++){
		if(wrd_tbl[i]==0){
			match=i+1;
			while(match<k){
				if(word[i]==word[match]){
					//printf("i is: %d, match is: %d, word[i] is: %c, word[match] is: %c \n", i, match, word[i], word[match]);
					wrd_tbl[match]=i;
				}
				match++;
			}
		}
	}
}

int main(int argc, char*argv[]){

	//GET THE WORD
	FILE * file_w =fopen("pattern.txt","r");
	fseek(file_w, 0L, SEEK_END);
	int word_len =ftell(file_w);
	rewind(file_w);
	char* word =(char*)malloc(word_len*sizeof(char));
	fgets(word, word_len, file_w);
	fclose(file_w);
	word_len=strlen(word);
	//printf("testing read: %c\n",word[0]);

	//GET THE TEXT FILE
	FILE * file_s =fopen("text_string.txt","r");	
	fseek(file_s, 0L, SEEK_END);
	int pat_len=ftell(file_s);
	rewind(file_s);
	char* text_string =(char*)malloc(pat_len*sizeof(char));
	fgets(text_string, pat_len, file_s);
	fclose(file_s);
	pat_len=strlen(text_string);
	//printf("testing pat: %s\n", text_string);
	
	//CREATE THE K-TABLE
	int* wrd_tbl= new int[word_len];
	for(int i=0;i<word_len;i++){
		wrd_tbl[i]=0;
	}
	tableBuild(word, wrd_tbl);


	//CALCULATIONS FOR BLOCK AND THREAD NUMBERS
	int threadnumber=pat_len/128;
	if(pat_len%128!=0){
		threadnumber++;
	}
	int blocknumber=1;
	if(threadnumber>1024){
		blocknumber=(threadnumber/1024);
		if(threadnumber%1024!=0){
			blocknumber++;
		}
		threadnumber=1024;
	}

	//PRINT INFORMATION
	printf("Word to find: <%s> - is a placeholder\n",word);
	printf("Wordlen is:  %d, Patlen is: %d\n",word_len, pat_len);
        printf("Array for ktable: ");
        for(int i=0;i<word_len;i++){
                printf("%d ",wrd_tbl[i]);
        }
        printf("\n");
	printf("Thread count: %d, Block count: %d\n", threadnumber, blocknumber);


	//
	//CPU TEST
	//

	//START CPU TIMING
	float cpu_time;
	cstart();

	//RUN TEST
	serial_alg(text_string, word, word_len, pat_len, wrd_tbl);

	//END CPU TIMING
	cend(&cpu_time);
	printf("CPU Serial time: %f\n", cpu_time);


	//
	//GPU TEST
	//

	//CREATE ALL OF THE CUDA VARIABLES
	char* d_text_string;
	char* d_word;
	int* d_wrd_tbl;
	int* d_ans;

	hipMalloc((void**)&d_text_string, pat_len*sizeof(char));
	hipMalloc((void**)&d_word, word_len*sizeof(char));
	hipMalloc((void**)&d_wrd_tbl, word_len*sizeof(int));
	hipMalloc((void**)&d_ans, pat_len*sizeof(int));

	hipMemcpy(d_text_string, text_string, pat_len*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_word, word, word_len*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_wrd_tbl, wrd_tbl, word_len*sizeof(int), hipMemcpyHostToDevice);

	//START GPU TIMING
	float gpu_time;
	gstart();
	
	//CALL CUDA KERNEL
	alg<<<blocknumber, threadnumber>>>(d_text_string, d_word, word_len, pat_len, d_wrd_tbl, d_ans);
	//TEST
	//alg<<<1,1>>>(d_text_string, d_word, word_len, pat_len, d_wrd_tbl);

	//RETRIEVE AND OUTPUT ANSWERS
	int *ans=new int[pat_len];
	hipMemcpy(ans, d_ans, pat_len*sizeof(int), hipMemcpyDeviceToHost);
	for(int i=0;i<pat_len;i++){
		if(ans[i]==1){
			printf("Pattern match found at position %d\n", i);
		}
		if(ans[i]==2){
			int ct=1, mtc=0;
			for(int j=i;j<i+(word_len-1);j++){
				if(text_string[j]==word[ct]){
					mtc++;
				}
				ct++;
			}
			if(mtc==(word_len-1)){
				printf("Pattern match found at position %d\n",i);
			}
		}

	}

	//END GPU TIMING
	gend(&gpu_time);
	printf("GPU Output time: %f\n", gpu_time);


	//FREE VARIABLES
	hipFree(d_text_string);
	hipFree(d_word);
	hipFree(d_wrd_tbl);
	hipFree(d_ans);
	free(text_string);
	free(word);
	free(wrd_tbl);
	free(ans);

	return 0;
}
